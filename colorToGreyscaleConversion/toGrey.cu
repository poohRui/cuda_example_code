#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define BLOCK_DIM 16
#define CHANNELS 3

__global__
void colorToGreyscaleConversion(unsigned char * Pout, 
                                unsigned char * Pin, 
                                int             width, 
                                int             height) {

    int Col = threadIdx.x + blockIdx.x * blockDim.x; 
    int Row = threadIdx.y + blockIdx.y * blockDim.y; 

    if (Col < width && Row < height) {
        int greyOffset = Row*width + Col;
        int rgbOffset = greyOffset*CHANNELS;
        unsigned char r = Pin[rgbOffset ]; 
        unsigned char g = Pin[rgbOffset+1]; 
        unsigned char b = Pin[rgbOffset+2]; 
        Pout[greyOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
    } 
}


void toGreyParallel(unsigned char * h_Pout, 
                    unsigned char * h_Pin, 
                    int             width, 
                    int             height){
    
    // Using device parallel calculate the result and finally print the time
    hipEvent_t start, stop;
    float elapsedTime = 0.0;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    unsigned char* d_Pout;
    unsigned char* d_Pin;

    size_t size_in = width * height * CHANNELS * sizeof(unsigned char);
    size_t size_out = width * height * sizeof(unsigned char);

    // Allocates object in the device global memory
    hipMalloc((void **)&d_Pout, size_out);
    hipMalloc((void **)&d_Pin, size_in);

    // Memory data transfer from host to device
    hipMemcpy(d_Pin, h_Pin, size_in, hipMemcpyHostToDevice);

    // Invoke kernel to do the computation on device
    dim3 dimGrid(ceil(width/(float)(BLOCK_DIM)),ceil(height/(float)(BLOCK_DIM)));
    dim3 dimBlock(BLOCK_DIM,BLOCK_DIM);
    colorToGreyscaleConversion<<<dimGrid, dimBlock>>>(d_Pout, d_Pin, width, height);

    // Transfer back the result from d_Pout to h_Pout
    hipMemcpy(h_Pout, d_Pout, size_out, hipMemcpyDeviceToHost);
    
    // Free device memory for Pout, Pin
    hipFree(d_Pout);
    hipFree(d_Pin);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    printf("Parallel invoke conversion function need %.1fs.\n",elapsedTime);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
}