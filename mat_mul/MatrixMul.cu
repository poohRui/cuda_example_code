#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

#define TILE_WIDTH 32

/**
 * This is a kernel MatrixMul function of parallel Matmul
 *
 * @param A    Matrix (m,dim)
 * @param B    Matrix (dim,n)
 * @param C    Result Matrix (m,n)
 * @param m      number of row in h_A
 * @param n      number of column in h_B
 * @param dim    number of row in h_B
 */
__global__
void MatrixMulKernel(float* A, 
                     float* B, 
                     float* C,
                     int    m, 
                     int    n, 
                     int    dim){

    __shared__ float Ads[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x; 
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Identify the row and column of the C element to work on
    // Each thread works on an element of C
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    // Calculate the number of phase
    int phase_num = ceil(dim / (float)TILE_WIDTH);

    bool cond0 = Row < m;
    bool cond1 = Col < n;

    float Cvalue = 0;
    // Each thread loads 'Row'th row of A and 'Col'th column of B
    for (int ph = 0; ph < phase_num; ++ph) {
        
        if(ph * TILE_WIDTH + tx < dim){
            Ads[ty][tx] = (cond0)?A[Row * dim + ph*TILE_WIDTH + tx]:0;   
        }
        else{
            Ads[ty][tx] = 0;
        }
        if(ph * TILE_WIDTH + ty < dim){
            Bds[ty][tx] = (cond1)?B[(ph*TILE_WIDTH + ty)*n + Col]:0;
        }
        else{
            Bds[ty][tx] = 0;
        }

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k) { 
            Cvalue += Ads[ty][k] * Bds[k][tx];
        }

        __syncthreads();
    }

    if(cond0 && cond1){
        C[Row * n + Col] = Cvalue;
    }
}

/**
 * This is a parallel Stub function of parallel Matmul
 *
 * @param h_A    Matrix (m,dim)
 * @param h_B    Matrix (dim,n)
 * @param h_C    Result Matrix (m,n)
 * @param m      number of row in h_A
 * @param n      number of column in h_B
 * @param dim    number of row in h_B
 */
void parallelMatMul(float* h_A, 
                    float* h_B, 
                    float* h_C, 
                    int    m, 
                    int    n, 
                    int    dim){

    // Using device parallel calculate the result and finally print the time
    hipEvent_t start, stop;
    float elapsedTime = 0.0;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    float *d_A, *d_B, *d_C;

    size_t size_of_float = sizeof(float);
    size_t size_A = m*dim*size_of_float;
    size_t size_B = n*dim*size_of_float;
    size_t size_C = m*n*size_of_float;

    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);
    
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);


    // Invoke kernel
    dim3 dimGrid(ceil(n/(float)(TILE_WIDTH)),ceil(m/(float)(TILE_WIDTH)),1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH,1);
    MatrixMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, m, n, dim);

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);
    
    // Free device memory for A, B, C
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Parallel invoke Matmul function need %.1fs.\n",elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

/**
 * This is a baseline kernel function of parallel Matmul
 *
 * @param A    Matrix (m,dim)
 * @param B    Matrix (dim,n)
 * @param C    Result Matrix (m,n)
 * @param m      number of row in h_A
 * @param n      number of column in h_B
 * @param dim    number of row in h_B
 */
__global__
void MatrixMulKernel_Baseline(float* A, 
                              float* B, 
                              float* C,
                              int    m, 
                              int    n, 
                              int    dim){
    
    int bx = blockIdx.x; 
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
                            
    // Identify the row and column of the C element to work on
    // Each thread works on an element of C
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    bool cond0 = Row < m;
    bool cond1 = Col < n;

    if(cond0 && cond1){
        float Cvalue = 0;
        for(int i = 0;i<dim;i++){
            Cvalue += A[Row*dim+i]*B[i*n+Col];
        }
        C[Row*n+Col] = Cvalue;
    }
}

/**
 * This is a baseline Parallel Stub function of parallel Matmul
 *
 * @param h_A    Matrix (m,dim)
 * @param h_B    Matrix (dim,n)
 * @param h_C    Result Matrix (m,n)
 * @param m      number of row in h_A
 * @param n      number of column in h_B
 * @param dim    number of row in h_B
 */
void parallelMatMul_baseline(float* h_A, 
                             float* h_B, 
                             float* h_C, 
                             int    m, 
                             int    n, 
                             int    dim){

    // Using device parallel calculate the result and finally print the time
    hipEvent_t start, stop;
    float elapsedTime = 0.0;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    float *d_A, *d_B, *d_C;

    size_t size_of_float = sizeof(float);
    size_t size_A = m*dim*size_of_float;
    size_t size_B = n*dim*size_of_float;
    size_t size_C = m*n*size_of_float;

    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);
    
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);


    // Invoke kernel
    dim3 dimGrid(ceil(n/(float)(TILE_WIDTH)),ceil(m/(float)(TILE_WIDTH)),1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH,1);
    MatrixMulKernel_Baseline<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, m, n, dim);

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);
    
    // Free device memory for A, B, C
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("The baseline Parallel invoke Matmul function need %.1fs.\n",elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}